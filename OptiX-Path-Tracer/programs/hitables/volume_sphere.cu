#include "hip/hip_runtime.h"
#include "../random.cuh"
#include "hitables.cuh"

/*! the parameters that describe each individual sphere geometry */
rtDeclareVariable(float3, center, , );
rtDeclareVariable(float, radius, , );
rtDeclareVariable(float, density, , );
rtDeclareVariable(int, index, , );

/*! the implicit state's ray we will intersect against */
rtDeclareVariable(Ray, ray, rtCurrentRay, );

/*! the attributes we use to communicate between intersection programs and hit
 * program */
rtDeclareVariable(HitRecord, hit_rec, attribute hit_rec, );

/*! the per ray data we operate on */
rtDeclareVariable(PerRayData, prd, rtPayload, );

RT_FUNCTION bool hit_boundary(const float tmin, const float tmax, float& rec) {
  const float3 oc = ray.origin - center;

  // if the ray hits the sphere, the following equation has two roots:
  // tdot(B, B) + 2tdot(B,A-C) + dot(A-C,A-C) - R = 0

  // Using Bhaskara's Formula, we have:
  const float a = dot(ray.direction, ray.direction);
  const float b = dot(oc, ray.direction);
  const float c = dot(oc, oc) - radius * radius;
  const float discriminant = b * b - a * c;

  // if the discriminant is lower than zero, there's no real
  // solution and thus no hit
  if (discriminant < 0.f) return false;

  // first root of the sphere equation:
  float temp = (-b - sqrtf(discriminant)) / a;

  // for a sphere, its normal is in (hitpoint - center)

  // if the first root was a hit,
  if (temp < tmax && temp > tmin) {
    rec = temp;
    return true;
  }

  // if the second root was a hit,
  temp = (-b + sqrtf(discriminant)) / a;
  if (temp < tmax && temp > tmin) {
    rec = temp;
    return true;
  }

  return false;
}

// Program that performs the ray-sphere intersection
//
// note that this is here is a simple, but not necessarily most numerically
// stable ray-sphere intersection variant out there. There are more
// stable variants out there, but for now let's stick with the one that
// the reference code used.
RT_PROGRAM void hit_sphere(int pid) {
  float rec1, rec2;

  if (hit_boundary(-FLT_MAX, FLT_MAX, rec1))
    if (hit_boundary(rec1 + 0.0001, FLT_MAX, rec2)) {
      if (rec1 < ray.tmin) rec1 = ray.tmin;

      if (rec2 > ray.tmax) rec2 = ray.tmax;

      if (rec1 >= rec2) return;

      if (rec1 < 0.f) rec1 = 0.f;

      float distance_inside_boundary = rec2 - rec1;
      distance_inside_boundary *= length(ray.direction);

      float hit_distance = -(1.f / density) * log(rnd(prd.seed));
      float temp = rec1 + hit_distance / length(ray.direction);

      if (rtPotentialIntersection(temp)) {
        hit_rec.distance = temp;

        hit_rec.view_direction = normalize(-ray.direction);

        float3 hit_point = ray.origin + temp * ray.direction;
        hit_point = rtTransformPoint(RT_OBJECT_TO_WORLD, hit_point);
        hit_rec.p = hit_point;

        float3 normal = make_float3(1.f, 0.f, 0.f);
        normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, normal));
        hit_rec.geometric_normal = normal;
        hit_rec.shading_normal = hit_rec.geometric_normal;

        hit_rec.u = 0.f;
        hit_rec.v = 0.f;

        hit_rec.index = index;

        rtReportIntersection(0);
      }
    }
}

/*! returns the bounding box of the pid'th primitive
  in this gometry. Since we only have one sphere in this
  program (we handle multiple spheres by having a different
  geometry per sphere), the'pid' parameter is ignored */
RT_PROGRAM void get_bounds(int pid, float result[6]) {
  Aabb* aabb = (Aabb*)result;
  aabb->m_min = center - radius;
  aabb->m_max = center + radius;
}
