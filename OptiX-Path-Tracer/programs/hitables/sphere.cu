#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "../prd.h"

/*! the parameters that describe each individual sphere geometry */
rtDeclareVariable(float3, center, , );
rtDeclareVariable(float,  radius, , );

/*! the implicit state's ray we will intersect against */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

/*! the attributes we use to communicate between intersection programs and hit program */
rtDeclareVariable(Hit_Record, hit_rec, attribute hit_rec, );

/*! the per ray data we operate on */
rtDeclareVariable(PerRayData, prd, rtPayload, );

inline __device__ void get_sphere_uv(const float3& p) {
	float phi = atan2(p.z, p.x);
	float theta = asin(p.y); 

	hit_rec.u = 1.f - (phi + PI_F) / (2.f * PI_F);
	hit_rec.v = (theta + PI_F / 2.f) / PI_F;
}

// Program that performs the ray-sphere intersection
//
// note that this is here is a simple, but not necessarily most numerically
// stable ray-sphere intersection variant out there. There are more
// stable variants out there, but for now let's stick with the one that
// the reference code used.
RT_PROGRAM void hit_sphere(int pid) {
  const float3 oc = ray.origin - center;

	// if the ray hits the sphere, the following equation has two roots:
	// tdot(B, B) + 2tdot(B,A-C) + dot(A-C,A-C) - R = 0

	// Using Bhaskara's Formula, we have:
  const float  a = dot(ray.direction, ray.direction);
  const float  b = dot(oc, ray.direction);
  const float  c = dot(oc, oc) - radius * radius;
  const float  discriminant = b * b - a * c;
  
  // if the discriminant is lower than zero, there's no real 
  // solution and thus no hit
  if (discriminant < 0.f) 
    return;

  // first root of the sphere equation:
  float temp = (-b - sqrtf(discriminant)) / a;

  // for a sphere, its normal is in (hitpoint - center)
  
  // if the first root was a hit,
  if (temp < ray.tmax && temp > ray.tmin) {
    if (rtPotentialIntersection(temp)) {
      hit_rec.distance = temp;

      float3 hit_point = ray.origin + temp * ray.direction;
      hit_point = rtTransformPoint(RT_OBJECT_TO_WORLD, hit_point);
      hit_rec.p = hit_point;

      float3 normal = (hit_rec.p - center) / radius;
      normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, normal));
      hit_rec.normal = normal;

      get_sphere_uv((hit_rec.p - center) / radius);

      hit_rec.index = 0;

      rtReportIntersection(0);
    }
  }
  
  // if the second root was a hit,
  temp = (-b + sqrtf(discriminant)) / a;
  if (temp < ray.tmax && temp > ray.tmin) {
    if (rtPotentialIntersection(temp)) {
      hit_rec.distance = temp;
      
      float3 hit_point = ray.origin + temp * ray.direction;
      hit_point = rtTransformPoint(RT_OBJECT_TO_WORLD, hit_point);
      hit_rec.p = hit_point;

      float3 normal = (hit_rec.p - center) / radius;
      normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, normal));
      hit_rec.normal = normal;

      get_sphere_uv((hit_rec.p - center) / radius);

      hit_rec.index = 0;

      rtReportIntersection(0);
    }
  }
}

/*! returns the bounding box of the pid'th primitive
  in this gometry. Since we only have one sphere in this 
  program (we handle multiple spheres by having a different
  geometry per sphere), the'pid' parameter is ignored */
RT_PROGRAM void get_bounds(int pid, float result[6]) {
  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->m_min = center - radius;
  aabb->m_max = center + radius;
}
