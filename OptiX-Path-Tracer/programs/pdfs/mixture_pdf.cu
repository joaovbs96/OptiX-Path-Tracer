#include "hip/hip_runtime.h"
#include "pdf.h"

rtDeclareVariable(rtCallableProgramId<float(PDFParams &)>, p0_value, , );
rtDeclareVariable(rtCallableProgramId<float(PDFParams &)>, p1_value, , );

RT_CALLABLE_PROGRAM float mixture_value(PDFParams &in) {
  return 0.5f * p0_value(in) + 0.5f * p1_value(in);
}

rtDeclareVariable(rtCallableProgramId<float3(PDFParams &, uint &)>, p0_generate,
                  , );
rtDeclareVariable(rtCallableProgramId<float3(PDFParams &, uint &)>, p1_generate,
                  , );

RT_CALLABLE_PROGRAM float3 mixture_generate(PDFParams &in, uint &seed) {
  if (rnd(seed) < 0.5f)
    return p0_generate(in, seed);
  else
    return p1_generate(in, seed);
}