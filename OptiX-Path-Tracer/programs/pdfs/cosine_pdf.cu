#include "hip/hip_runtime.h"
#include "pdf.h"

RT_CALLABLE_PROGRAM float3 cosine_generate(pdf_in &in, XorShift32 &rnd) {
    in.uvw.build_from_w(in.normal);
    
    float3 temp = random_cosine_direction(rnd);
    in.scattered_direction = in.uvw.local(temp);
    
    return in.scattered_direction;
}

RT_CALLABLE_PROGRAM float cosine_value(pdf_in &in) {
    float cosine = dot(unit_vector(in.scattered_direction), in.uvw.w);
    
    if(cosine > 0.f)
        return cosine / PI_F;
    else
        return 0.f;
}