// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "materials/material.h"

// the implicit state's ray we will intersect against
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
// the per ray data we operate on
rtDeclareVariable(PerRayData, prd, rtPayload, );

RT_PROGRAM void sky() {
  const vec3f unit_direction = normalize(ray.direction);
  const float t = 0.5f*(unit_direction.y + 1.0f);
  const vec3f c = (1.0f - t) * vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
  prd.out.attenuation = c;
  prd.out.scatterEvent = rayDidntHitAnything;
}

RT_PROGRAM void dark() {
  prd.out.attenuation = vec3f(0.f);
  prd.out.scatterEvent = rayDidntHitAnything;
}
