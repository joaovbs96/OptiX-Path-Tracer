#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "material.h"

// the implicit state's ray we will intersect against
rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );

// the per ray data we operate on
rtDeclareVariable(PerRayData, prd,   rtPayload, );
rtDeclareVariable(rtObject,   world, , );

// the attributes we use to communicate between intersection programs and hit program
rtDeclareVariable(Hit_Record, hit_rec, attribute hit_rec, );

// and finally - that particular material's parameters
rtBuffer< rtCallableProgramId<float3(float, float, float3)> > sample_texture; // no need to use this here
rtDeclareVariable(float, ref_idx, , );


inline __device__ bool scatter(const optix::Ray &ray_in) {
  prd.out.is_specular = true;
  prd.out.origin = hit_rec.p;
  prd.out.attenuation = make_float3(1.f);
  prd.out.normal = hit_rec.normal;
  
  float3 outward_normal;
  float ni_over_nt;
  float cosine;
  if (dot(ray_in.direction, hit_rec.normal) > 0.f) {
    outward_normal = -1 * hit_rec.normal;
    ni_over_nt = ref_idx;
    cosine = ref_idx * dot(ray_in.direction, hit_rec.normal) / length(ray_in.direction);
  }
  else {
    outward_normal = hit_rec.normal;
    ni_over_nt = 1.f / ref_idx;
    cosine = -dot(ray_in.direction, hit_rec.normal) / length(ray_in.direction);
  }
  
  float3 refracted;
  float reflect_prob;
  if (refract(ray_in.direction, outward_normal, ni_over_nt, refracted)) 
    reflect_prob = schlick(cosine, ref_idx);
  else 
    reflect_prob = 1.f;

  float3 reflected = reflect(ray_in.direction, hit_rec.normal);
  if ((*prd.in.randState)() < reflect_prob) 
    prd.out.direction = reflected;
  else 
    prd.out.direction = refracted;
  
  return true;
}

inline __device__ float3 emitted() {
  return make_float3(0.f, 0.f, 0.f);
}

RT_PROGRAM void closest_hit() {
  prd.out.type = Dielectric;
  prd.out.emitted = emitted();
  prd.out.scatterEvent = scatter(ray) ? rayGotBounced : rayGotCancelled;
}
