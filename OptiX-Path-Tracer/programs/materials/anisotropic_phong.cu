#include "hip/hip_runtime.h"

#include "material.cuh"
#include "microfacets.cuh"

////////////////////////////////////////////////////////////
// --- Ashikhmin-Shirley Anisotropic Phong BRDF Model --- //
////////////////////////////////////////////////////////////

// Original Paper & Tech Report - "An Anisotropic Phong Light Reflection Model"
// https://www.cs.utah.edu/~shirley/papers/jgtbrdf.pdf
// https://www.cs.utah.edu/docs/techreports/2000/pdf/UUCS-00-014.pdf

// Reference Implementation:
// https://developer.blender.org/diffusion/C/browse/master/src/kernel/closure/bsdf_ashikhmin_shirley.h
// FresnelBlend from PBRT
// https://github.com/mmp/pbrt-v3/blob/9f717d847a807793fa966cf0eaa366852efef167/src/core/reflection.cpp
// https://github.com/mmp/pbrt-v3/blob/9f717d847a807793fa966cf0eaa366852efef167/src/core/reflection.h

// OptiX Context objects
rtDeclareVariable(Ray, ray, rtCurrentRay, );                 // current ray
rtDeclareVariable(PerRayData, prd, rtPayload, );             // ray PRD
rtDeclareVariable(HitRecord, hit_rec, attribute hit_rec, );  // from geometry

// Material Parameters
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3, int)>,
                  diffuse_color, , );
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3, int)>,
                  specular_color, , );
rtDeclareVariable(float, nu, , );
rtDeclareVariable(float, nv, , );

///////////////////////////
// --- BRDF Programs --- //
///////////////////////////

// Assigns material and hit parameters to PRD
RT_PROGRAM void closest_hit() {
  prd.matType = Anisotropic_Material;
  prd.isSpecular = false;
  prd.scatterEvent = rayGotBounced;

  // Get hit params
  prd.origin = hit_rec.p;
  prd.geometric_normal = hit_rec.geometric_normal;
  prd.shading_normal = hit_rec.shading_normal;

  // Get material colors
  int index = hit_rec.index;
  float3 diffuse = diffuse_color(hit_rec.u, hit_rec.v, hit_rec.p, index);
  float3 specular = specular_color(hit_rec.u, hit_rec.v, hit_rec.p, index);

  // Assign material parameters to PRD, to be used in the sampling programs
  prd.matParams.anisotropic.diffuse_color = diffuse;
  prd.matParams.anisotropic.specular_color = specular;
  prd.matParams.anisotropic.nu = Beckmann_Roughness(nu);
  prd.matParams.anisotropic.nv = Beckmann_Roughness(nv);
}

// Samples BRDF, generating outgoing direction(Wo)
RT_CALLABLE_PROGRAM float3 BRDF_Sample(PDFParams &pdf, uint &seed) {
  // Get material params from input variable
  float nu = pdf.matParams.anisotropic.nu;
  float nv = pdf.matParams.anisotropic.nv;

  // random variables
  float2 random = make_float2(rnd(seed), rnd(seed));

  float3 direction;
  if (random.x < 0.5) {
    random.x = min(2 * random.x, 1.f - 1e-6f);

    // Cosine-sample the hemisphere, flipping the direction if necessary
    cosine_sample_hemisphere(random.x, random.y, direction);

    Onb uvw(pdf.normal);
    uvw.inverse_transform(direction);

    if (!SameHemisphere(pdf.origin, direction)) direction *= -1.f;

  } else {
    random.x = min(2 * (random.x - 0.5f), 1.f - 1e-6f);

    // Sample microfacet orientation(H) and reflected direction(origin)
    float3 H = Beckmann_Sample(pdf.origin, random, nu, nv);
    direction = -reflect(pdf.origin, H);  // TODO: or -reflect

    if (!SameHemisphere(pdf.origin, direction)) direction = make_float3(0.f);
  }

  pdf.direction = direction;

  return pdf.direction;
}

// Gets BRDF PDF value
RT_CALLABLE_PROGRAM float BRDF_PDF(PDFParams &pdf) {
  if (!SameHemisphere(pdf.origin, pdf.direction)) return 0.f;

  // Get material params from input variable
  float nu = pdf.matParams.anisotropic.nu;
  float nv = pdf.matParams.anisotropic.nv;

  // half vector = (v1 + v2) / |v1 + v2|
  float3 H = normalize(pdf.direction + pdf.origin);
  float H_PDF = Beckmann_PDF(H, nu, nv);
  float HdotI = dot(H, pdf.origin);

  float AbsCosThetaWo = AbsCosTheta(pdf.direction);

  return 0.5f * (AbsCosThetaWo * (1.f / PI_F) + H_PDF / (4.f * HdotI));
}

// Evaluates BRDF, returning its reflectance
RT_CALLABLE_PROGRAM float3 BRDF_Evaluate(PDFParams &pdf) {
  // Get material params from input variable
  float3 Rd, Rs;
  Rd = pdf.matParams.anisotropic.diffuse_color;
  Rs = pdf.matParams.anisotropic.specular_color;
  float nu = pdf.matParams.anisotropic.nu;
  float nv = pdf.matParams.anisotropic.nv;

  float nk1 = AbsCosTheta(pdf.origin);     // wo - origin
  float nk2 = AbsCosTheta(pdf.direction);  // wi - direction

  // diffuse component
  float3 diffuse_component = Rd * (28.f / (23.f * PI_F));
  diffuse_component *= (make_float3(1.f) - Rs);
  diffuse_component *= (1.f - powf(1.f - nk1 * 0.5f, 5.f));
  diffuse_component *= (1.f - powf(1.f - nk2 * 0.5f, 5.f));

  // half vector = (v1 + v2) / |v1 + v2|
  float3 H = normalize(pdf.direction + pdf.origin);
  if (isNull(H)) return make_float3(0.f);
  float HdotO = dot(H, pdf.direction);

  // specular component
  float3 specular_component = schlick(Rs, HdotO);  // fresnel reflectance
  specular_component *= Beckmann_D(H, nu, nv);  // microfacet distribution term
  specular_component /= (4.f * abs(HdotO) * ffmax(nk1, nk2));

  return (diffuse_component + specular_component) * nk2;
}