#include "hip/hip_runtime.h"

#include "material.cuh"

// Paper & Tech Report
// https://www.cs.utah.edu/~shirley/papers/jgtbrdf.pdf
// https://www.cs.utah.edu/docs/techreports/2000/pdf/UUCS-00-014.pdf

// reference:
// https://github.com/JerryCao1985/SORT/blob/master/src/bsdf/ashikhmanshirley.cpp
// https://github.com/JerryCao1985/SORT/blob/master/src/bsdf/ashikhmanshirley.h

// origin -> k1
// direction -> k2
// N -> normal
// Rd -> diffuse color(of the 'substrate' under the specular coating)
// Rs -> specular color
// nu, nv -> phong parameters

// OptiX Context objects
rtDeclareVariable(Ray, ray, rtCurrentRay, );                 // current ray
rtDeclareVariable(PerRayData, prd, rtPayload, );             // ray PRD
rtDeclareVariable(HitRecord, hit_rec, attribute hit_rec, );  // from geometry

// Material Parameters
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3, int)>,
                  diffuse_color, , );
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3, int)>,
                  specular_color, , );
rtDeclareVariable(float, nu, , );
rtDeclareVariable(float, nv, , );

// Material Programs
RT_PROGRAM void closest_hit() {
  prd.matType = Anisotropic_Material;
  prd.isSpecular = false; // this parameter refers to 'ideal specular BRDFs', like Metal and Dielectric
  prd.scatterEvent = rayGotBounced;

  prd.origin = hit_rec.p;
  prd.geometric_normal = hit_rec.geometric_normal;
  prd.shading_normal = hit_rec.shading_normal;

  // Get Material Colors
  int index = hit_rec.index;
  float3 diffuse = diffuse_color(hit_rec.u, hit_rec.v, hit_rec.p, index);
  float3 specular = specular_color(hit_rec.u, hit_rec.v, hit_rec.p, index);

  // Assign material parameters to PRD, to be used in the BRDF programs
  MaterialParameters params;
  params.u = hit_rec.u;
  params.v = hit_rec.v;
  params.anisotropic.nu = nu;
  params.anisotropic.nv = nv;
  params.anisotropic.diffuse_color = diffuse;
  params.anisotropic.specular_color = specular;
  prd.matParams = params;
}

RT_CALLABLE_PROGRAM float3 BRDF_Sample(PDFParams &pdf, uint &seed) {
  // Get material params from input variable
  MaterialParameters param = pdf.matParams;
  float nu = param.anisotropic.nu;
  float nv = param.anisotropic.nv;
  float u = rnd(seed);
  float v = rnd(seed);

  float3 direction;
  if (u < 0.5f) {
    cosine_sample_hemisphere(u, v, direction);

    Onb uvw(pdf.normal);
    uvw.inverse_transform(direction);
  } else {
    direction = Blinn_Sample(u, v, nu, nv);
    direction = 2 * dot(pdf.origin, direction) * direction - pdf.origin;
  }

  pdf.direction = direction;

  return pdf.direction;
}

// TODO: check microfacet BRDF description in the SIGGRAPH courses and the following link
// http://simonstechblog.blogspot.com/2011/12/microfacet-brdf.html

// TODO: check blinn functions, they are most likely returning the NaNs

RT_CALLABLE_PROGRAM float BRDF_PDF(PDFParams &pdf) {
  // Get material params from input variable
  MaterialParameters param = pdf.matParams;
  float nu = param.anisotropic.nu;
  float nv = param.anisotropic.nv;

  // half vector = (v1 + v2) / |v1 + v2|
  float3 half_vector = unit_vector(pdf.origin + pdf.direction);
  //printf("%f %f %f \n", half_vector.x, half_vector.y, half_vector.z);
  float h_pdf = Blinn_PDF(half_vector, nu, nv);
  h_pdf = ffmax(0.001f, h_pdf);

  float a = AbsCosTheta(pdf.origin) / PI_F;
  float b = h_pdf / (4.f * dot(pdf.direction, half_vector));
  float t = 0.5f;

  // FIXME: it's returning NaN

  return ffmax(0.001f, lerp(a, b, t));
}

RT_CALLABLE_PROGRAM float3 BRDF_Evaluate(PDFParams &pdf) {
  // Get material params from input variable
  MaterialParameters param = pdf.matParams;
  float3 diffuse_color = param.anisotropic.diffuse_color;
  float3 specular_color = param.anisotropic.specular_color;
  float nu = param.anisotropic.nu;
  float nv = param.anisotropic.nv;

  float nk1 = AbsCosTheta(pdf.origin);     // cos_theta_i
  float nk2 = AbsCosTheta(pdf.direction);  // cos_theta_o

  // diffuse component
  float3 diffuse_component = 28.f * diffuse_color;
  diffuse_component /= 23.f * PI_F;
  diffuse_component *= (make_float3(1.f) - specular_color);
  diffuse_component *= (1.f - fresnel_schlick(nk1 / 2));
  diffuse_component *= (1.f - fresnel_schlick(nk2 / 2));

  // half vector = (v1 + v2) / |v1 + v2|
  float3 half_vector = unit_vector(pdf.origin + pdf.direction);

  // specular component
  float IoH = dot(pdf.origin, unit_vector(half_vector));
  printf("%f\n", IoH);
  float3 specular_component = schlick(specular_color, IoH);
  specular_component *= Blinn_Density(half_vector, nu, nv);
  specular_component /= 4.f * IoH * ffmax(nk1, nk2);

  // FIXME: Specular is returning NaN

  return (diffuse_component + specular_component) * nk1;
}