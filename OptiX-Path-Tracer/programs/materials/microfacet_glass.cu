#include "hip/hip_runtime.h"
#include "light_sample.cuh"

////////////////////////////////////
// --- Microfacet Glass Model --- //
////////////////////////////////////

// Based on PBRT code & theory
// http://www.pbr-book.org/3ed-2018/Reflection_Models/Microfacet_Models.html#TheTorrancendashSparrowModel
// https://github.com/mmp/pbrt-v3/blob/9f717d847a807793fa966cf0eaa366852efef167/src/core/reflection.h#L429

// OptiX Context objects
rtDeclareVariable(Ray, ray, rtCurrentRay, );                 // current ray
rtDeclareVariable(PerRayData, prd, rtPayload, );             // ray PRD
rtDeclareVariable(rtObject, world, , );                      // scene graph
rtDeclareVariable(HitRecord, hit_rec, attribute hit_rec, );  // from geometry

// Material Parameters
rtDeclareVariable(Texture_Function, sample_texture, , );
rtDeclareVariable(float, nu, , );
rtDeclareVariable(float, nv, , );

RT_FUNCTION Torrance_Sparrow_Parameters Get_Parameters(const float3 &P, float u,
                                                       float v, int index) {
  Torrance_Sparrow_Parameters surface;

  surface.color = sample_texture(u, v, P, index);
  surface.nu = nu;
  surface.nv = nv;

  return surface;
}

// Assigns material and hit parameters to PRD
RT_PROGRAM void closest_hit() {
  int index = hit_rec.index;
  float u = hit_rec.u, v = hit_rec.v;
  float3 P = hit_rec.p, Wo = hit_rec.view_direction;
  float3 N = hit_rec.shading_normal;

  Torrance_Sparrow_Parameters surface = Get_Parameters(P, u, v, index);

  // Sample BRDF
  float3 Wi = Sample(surface, P, Wo, N, prd.seed);
  float pdf;  // calculated in the Evaluate function
  float3 attenuation = Evaluate(surface, P, Wo, Wi, N, pdf);

  // Assign parameters to PRD
  prd.scatterEvent = rayGotBounced;
  prd.origin = hit_rec.p;
  prd.direction = Wi;
  prd.throughput *= clamp(attenuation / pdf, 0.f, 1.f);
  prd.isSpecular = true;
}

// Evaluates BRDF, returning its reflectance
RT_CALLABLE_PROGRAM float3 BRDF_Evaluate(PDFParams &pdf) {
  float3 Wo = pdf.view_direction, Wi = pdf.direction;
  float3 Rs = pdf.matParams.attenuation;
  float nu = pdf.matParams.anisotropic.nu;
  float nv = pdf.matParams.anisotropic.nv;
  
  float cosThetaI = CosTheta(Wi), cosThetaO = CosTheta(Wo);
  if (cosThetaI == 0.f || cosThetaO == 0.f) return make_float3(0.f);

  float eta = CosTheta(Wo) > 0 ? (nv / nu) : (nu / nv);
  float3 H = normalize(Wo + Wi * eta);
  if(H.y < 0) H *= -1;

  // *Note that this is not a symetric BRDF. The PBRT implementation take both 
  // directions into account. The code here works only on unidirectional PTs.
  // https://github.com/mmp/pbrt-v3/blob/f7653953b2f9cc5d6a53b46acb5ce03317fd3e8b/src/core/reflection.cpp#L260

  float HdotO = dot(H, Wo), HdotI = dot(H, Wi);
  float AHdotO = fabsf(HdotO), AHdotI = fabsf(HdotI);

  float denom = Square(HdotO + eta * HdotI) * cosThetaI * cosThetaO;

  float3 F = schlick(Rs, HdotI);    // Fresnel Reflectance
  float G = GGX_G(Wo, Wi, nu, nv);  // Geometric Shadowing
  float D = GGX_D(H, nu, nv);       // Normal Distribution Function(NDF)

  return (Rs * (make_float3(1.f) - F) * D * G  * AHdotI * AHdotO) / denom;
}