#include "hip/hip_runtime.h"
#include "disney.cuh"

////////////////////////////////////
// --- Disney Principled BSDF --- //
////////////////////////////////////

// Based on blog post and code by Joe Schutte
// https://schuttejoe.github.io/post/disneybsdf/
// https://github.com/schuttejoe/Selas/blob/dev/Source/Core/Shading/Disney.cpp
// https://github.com/schuttejoe/Selas/blob/dev/Source/Core/Shading/Disney.h

// OptiX Context objects
rtDeclareVariable(Ray, ray, rtCurrentRay, );                 // current ray
rtDeclareVariable(PerRayData, prd, rtPayload, );             // ray PRD
rtDeclareVariable(HitRecord, hit_rec, attribute hit_rec, );  // from geometry

// Material Parameters
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3, int)>,
                  BaseColor, , );
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3, int)>,
                  TransmittanceColor, , );
rtDeclareVariable(float, nu, , );
rtDeclareVariable(float, nv, , );

///////////////////////////
// --- BRDF Programs --- //
///////////////////////////

// Assigns material and hit parameters to PRD
RT_PROGRAM void closest_hit() {
  prd.matType = Torrance_Sparrow_BRDF;
  prd.isSpecular = false;
  prd.scatterEvent = rayGotBounced;

  // Get hit params
  prd.origin = hit_rec.p;
  prd.geometric_normal = normalize(hit_rec.geometric_normal);
  prd.shading_normal = normalize(hit_rec.shading_normal);
  prd.view_direction = normalize(hit_rec.view_direction);

  // Get material color
  int index = hit_rec.index;
  float3 color = sample_texture(hit_rec.u, hit_rec.v, hit_rec.p, index);

  // Assign material parameters to PRD, to be used in the sampling programs
  prd.matParams.attenuation = color;
  prd.matParams.anisotropic.nu = nu;
  prd.matParams.anisotropic.nv = nv;
}

// Samples BRDF, generating outgoing direction(Wo)
RT_CALLABLE_PROGRAM float3 BRDF_Sample(PDFParams &pdf, uint &seed) {
  // Get material params from input variable
  float nu = pdf.matParams.anisotropic.nu;
  float nv = pdf.matParams.anisotropic.nv;

  float3 Wo = pdf.view_direction;  // outgoing, to camera

  // create basis
  float3 N = normalize(pdf.geometric_normal);
  float3 T = normalize(cross(N, make_float3(0.f, 1.f, 0.f)));
  float3 B = cross(T, N);

  // random variables
  float2 random = make_float2(rnd(seed), rnd(seed));
  pdf.matParams.u = random.x;
  pdf.matParams.v = random.y;

  // get half vector and rotate it to world space
  float3 H = normalize(GGX_Sample(Wo, random, nu, nv));
  H = H.x * B + H.y * N + H.z * T;

  float HdotI = dot(H, Wo);
  if (HdotI < 0.f) H = -H;

  float3 Wi = normalize(-Wo + 2.f * dot(Wo, H) * H);  // reflect(Wo, H)

  pdf.direction = Wi;

  return pdf.direction;
}

// Gets BRDF PDF value
RT_CALLABLE_PROGRAM float BRDF_PDF(PDFParams &pdf) {
  float3 Wo = pdf.view_direction;
  float3 Wi = pdf.localDirection;

  // Get material params from input variable
  float nu = pdf.matParams.anisotropic.nu;
  float nv = pdf.matParams.anisotropic.nv;

  // Handles degenerate cases for microfacet reflection
  float3 H = normalize(Wi + Wo);

  return GGX_PDF(H, Wo, nu, nv) / (4.f * dot(Wo, H));
}

// Evaluates BRDF, returning its reflectance
RT_CALLABLE_PROGRAM float3 BRDF_Evaluate(PDFParams &pdf) {
  float3 Wo = pdf.view_direction, Wi = pdf.direction;
  // float3 wo = Normalize(MatrixMultiply(v, surface.worldToTangent));
  // float3 wi = Normalize(MatrixMultiply(l, surface.worldToTangent));
  float3 H = normalize(Wo + Wi);

  // Get material params from input variable
  Disney_Parameters surface = pdf.matParams.disney;

  float dotNV = CosTheta(Wo);
  float dotNL = CosTheta(Wi);

  float3 reflectance = make_float3(0.f);

  // TODO: implement pdf functions

  float3 baseColor = surface.baseColor;
  float metallic = surface.metallic;
  float specTrans = surface.specTrans;
  float roughness = surface.roughness;

  // calculate all of the anisotropic params
  float ax, ay;
  Disney_Anisotropic_Params(surface.roughness, surface.anisotropic, ax, ay);

  float diffuseWeight = (1.f - metallic) * (1.f - specTrans);
  float transWeight = (1.f - metallic) * specTrans;

  // Clearcoat
  bool upperHemisphere = dotNL > 0.f && dotNV > 0.f;
  if (upperHemisphere && surface.clearcoat > 0.f) {
    float clearcoat = EvaluateDisneyClearcoat(surface, Wo, H, Wi);
    reflectance += float3(clearcoat);
  }

  // Diffuse
  if (diffuseWeight > 0.f) {
    float diffuse = EvaluateDisneyDiffuse(surface, Wo, H, Wi, thin);
    float3 sheen = EvaluateSheen(surface, Wo, H, Wi);

    reflectance += diffuseWeight * (diffuse * surface.baseColor + sheen);
  }

  // Transmission
  if (transWeight > 0.f) {
    // Scale roughness based on IOR (Burley 2015, Figure 15).
    float rscaled =
        thin ? ThinTransmissionRoughness(surface.ior, surface.roughness)
             : surface.roughness;
    float tax, tay;
    Disney_Anisotropic_Params(rscaled, surface.anisotropic, tax, tay);

    float3 transmission =
        EvaluateDisneySpecTransmission(surface, Wo, H, Wi, tax, tay, thin);
    reflectance += transWeight * transmission;
  }

  // -- specular
  if (upperHemisphere) {
    float3 specular = EvaluateDisneyBRDF(
        surface, wo, wm, wi, forwardMetallicPdfW, reverseMetallicPdfW);

    reflectance += specular;
  }

  reflectance = reflectance * Absf(dotNL);

  return reflectance;
}