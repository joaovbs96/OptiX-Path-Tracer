#include "hip/hip_runtime.h"
#include "disney.cuh"

// TODO: merge PDf programs into the evaluate programs(prevents redoing work)
// TODO: refactor the Eval and sample programs into functions
// TODO: refactor the Light Sample program into a templated function with
// material parameters

////////////////////////////////////
// --- Disney Principled BSDF --- //
////////////////////////////////////

// Based on blog post and code by Joe Schutte
// https://schuttejoe.github.io/post/disneybsdf/
// https://github.com/schuttejoe/Selas/blob/dev/Source/Core/Shading/Disney.cpp
// https://github.com/schuttejoe/Selas/blob/dev/Source/Core/Shading/Disney.h

// Note that the microfacet functions used here are the variations implemented
// in the disney.cuh file, rather than the microfacet.cuh header used by other
// BRDF models.

// OptiX Context objects
rtDeclareVariable(Ray, ray, rtCurrentRay, );                 // current ray
rtDeclareVariable(PerRayData, prd, rtPayload, );             // ray PRD
rtDeclareVariable(HitRecord, hit_rec, attribute hit_rec, );  // from geometry

// Material Parameters
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3, int)>,
                  BaseColor, , );
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3, int)>,
                  TransmittanceColor, , );
rtDeclareVariable(float, nu, , );
rtDeclareVariable(float, nv, , );

///////////////////////////
// --- BRDF Programs --- //
///////////////////////////

// Assigns material and hit parameters to PRD
RT_PROGRAM void closest_hit() {
  prd.matType = Torrance_Sparrow_BRDF;
  prd.isSpecular = false;
  prd.scatterEvent = rayGotBounced;

  // Get hit params
  prd.origin = hit_rec.p;
  prd.geometric_normal = normalize(hit_rec.geometric_normal);
  prd.shading_normal = normalize(hit_rec.shading_normal);
  prd.view_direction = normalize(hit_rec.view_direction);

  // Get material color
  int index = hit_rec.index;
  float3 color = sample_texture(hit_rec.u, hit_rec.v, hit_rec.p, index);

  // Assign material parameters to PRD, to be used in the sampling programs
  prd.matParams.attenuation = color;
  prd.matParams.anisotropic.nu = nu;
  prd.matParams.anisotropic.nv = nv;
}

// Samples BRDF, generating outgoing direction(Wo)
RT_CALLABLE_PROGRAM float3 BRDF_Sample(PDFParams &pdf, uint &seed) {
  // Get material params from input variable
  float nu = pdf.matParams.anisotropic.nu;
  float nv = pdf.matParams.anisotropic.nv;

  float3 Wo = pdf.view_direction;  // outgoing, to camera

  // create basis
  float3 N = normalize(pdf.geometric_normal);
  float3 T = normalize(cross(N, make_float3(0.f, 1.f, 0.f)));
  float3 B = cross(T, N);

  // random variables
  float2 random = make_float2(rnd(seed), rnd(seed));
  pdf.matParams.u = random.x;
  pdf.matParams.v = random.y;

  // get half vector and rotate it to world space
  float3 H = normalize(GGX_Sample(Wo, random, nu, nv));
  H = H.x * B + H.y * N + H.z * T;

  float HdotI = dot(H, Wo);
  if (HdotI < 0.f) H = -H;

  float3 Wi = normalize(-Wo + 2.f * dot(Wo, H) * H);  // reflect(Wo, H)

  pdf.direction = Wi;

  return pdf.direction;
}

// Gets BRDF PDF value
RT_CALLABLE_PROGRAM float BRDF_PDF(PDFParams &pdf) {
  // PDF is computed on evaluate function
  return 1.f;
}

// Evaluates BRDF, returning its reflectance
RT_CALLABLE_PROGRAM float3 BRDF_Evaluate(PDFParams &pdf) {
  float3 Wo = pdf.view_direction, Wi = pdf.direction;
  // TODO: check worldToTangent transform
  // float3 wo = Normalize(MatrixMultiply(v, surface.worldToTangent));
  // float3 wi = Normalize(MatrixMultiply(l, surface.worldToTangent));
  float3 H = normalize(Wo + Wi);

  // Get material params from input variable
  Disney_Parameters surface = pdf.matParams.disney;

  float dotNV = CosTheta(Wo);
  float dotNL = CosTheta(Wi);

  float3 reflectance = make_float3(0.f);

  float disneyPdf = 0.f;                          // 'final' PDF value
  float pBRDF, pDiffuse, pClearcoat, pSpecTrans;  // PDF weights of each lobe
  CalculateLobePdfs(surface, pBRDF, pDiffuse, pClearcoat, pSpecTrans);

  float3 baseColor = surface.baseColor;
  float metallic = surface.metallic;
  float specTrans = surface.specTrans;
  float roughness = surface.roughness;

  // calculate all of the anisotropic params
  float ax, ay;
  Anisotropic_Params(surface.roughness, surface.anisotropic, ax, ay);

  float diffuseWeight = (1.f - metallic) * (1.f - specTrans);
  float transWeight = (1.f - metallic) * specTrans;

  // Clearcoat
  bool upperHemisphere = dotNL > 0.f && dotNV > 0.f;
  if (upperHemisphere && surface.clearcoat > 0.f) {
    float clearcoatPdf;
    float clearcoat = Evaluate_Clearcoat(surface, Wo, H, Wi, clearcoatPdf);

    reflectance += make_float3(clearcoat);
    disneyPdf += pClearcoat * clearcoatPdf;
  }

  // Diffuse
  if (diffuseWeight > 0.f) {
    float diffusePdf = AbsCosTheta(wi);
    float diffuse = Evaluate_Diffuse(surface, Wo, H, Wi, thin);
    float3 sheen = Evaluate_Sheen(surface, Wo, H, Wi);

    reflectance += diffuseWeight * (diffuse * surface.baseColor + sheen);
    disneyPdf += pDiffuse * diffusePdf;
  }

  // Transmission(Refraction)
  if (transWeight > 0.f) {
    float rscaled;
    if (thin)  // Scale roughness based on IOR (Burley 2015, Figure 15).
      rscaled = Transmission_Roughness(surface.ior, surface.roughness);
    else
      rscaled = surface.roughness;

    float tax, tay;
    Anisotropic_Params(rscaled, surface.anisotropic, tax, tay);

    // TODO: probably missing something here for refraction
    // TODO: implement microfacet glass model

    float3 transmission =
        Evaluate_Transmission(surface, Wo, H, Wi, tax, tay, thin);
    reflectance += transWeight * transmission;

    float transmissivePdf = GGX_PDF(Wi, H, Wo, tax, tay);

    transmissivePdf /= Square(dot(H, Wi) + surface.relativeIOR * dot(H, Wo));
    disneyPdf += pSpecTrans * transmissivePdf;
  }

  // -- specular
  if (upperHemisphere) {
    float metallicPdf;
    float3 specular = Evaluate_Specular(surface, Wo, H, Wi, metallicPdf);

    reflectance += specular;
    disneyPdf += pBRDF * metallicPdf / (4 * fabsf(dot(Wo, H)));
  }

  reflectance *= fabsf(dotNL);

  return reflectance / disneyPdf;
}