#include "hip/hip_runtime.h"
#include "material.h"

/*! the implicit state's ray we will intersect against */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

/*! the per ray data we operate on */
rtDeclareVariable(PerRayData, prd, rtPayload, );
rtDeclareVariable(rtObject, world, , );

/*! the attributes we use to communicate between intersection programs and hit program */
rtDeclareVariable(float3, hit_rec_normal, attribute hit_rec_normal, );
rtDeclareVariable(float3, hit_rec_p, attribute hit_rec_p, );
rtDeclareVariable(float, hit_rec_u, attribute hit_rec_u, );
rtDeclareVariable(float, hit_rec_v, attribute hit_rec_v, );

/*! and finally - that particular material's parameters */
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3)>, sample_texture, , );


/*! the actual scatter function - in Pete's reference code, that's a
  virtual function, but since we have a different function per program
  we do not need this here */
  inline __device__ bool scatter(const optix::Ray &ray_in,
                                 DRand48 &rndState,
                                 vec3f &scattered_origin,
                                 vec3f &scattered_direction,
                                 vec3f &attenuation,
                                 float &pdf) {
  return false;
}

inline __device__ float scattering_pdf(){
  return false;
}

inline __device__ float3 emitted(){
  if(dot(hit_rec_normal, ray.direction) < 0.f)
    return sample_texture(hit_rec_u, hit_rec_v, hit_rec_p);
  else
    return make_float3(0.f);
}

RT_PROGRAM void closest_hit() {
  prd.out.emitted = emitted();
  prd.out.normal = hit_rec_normal;
  prd.out.scatterEvent
    = scatter(ray,
              *prd.in.randState,
              prd.out.scattered_origin,
              prd.out.scattered_direction,
              prd.out.attenuation,
              prd.out.pdf)
    ? rayGotBounced
    : rayGotCancelled;
  prd.out.scattered_pdf = scattering_pdf();
}
