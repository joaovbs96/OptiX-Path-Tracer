#include "hip/hip_runtime.h"
#include "material.cuh"

//////////////////////////////////////////
// --- Oren-Nayar Reflectance Model --- //
//////////////////////////////////////////

// Original Paper: "Generalization of Lambert’s Reflectance Model"
// http://www.cs.columbia.edu/CAVE/projects/oren/
// http://www1.cs.columbia.edu/CAVE/publications/pdfs/Oren_SIGGRAPH94.pdf

// Yasuhiro Fujii’s "A tiny improvement of Oren-Nayar reflectance model" variant
// http://mimosa-pudica.net/improved-oren-nayar.html

// Reference Implementations:
// https://developer.blender.org/diffusion/C/browse/master/src/kernel/closure/bsdf_oren_nayar.h
// https://github.com/mmp/pbrt-v3/blob/f7653953b2f9cc5d6a53b46acb5ce03317fd3e8b/src/core/reflection.cpp#L197-L224

// OptiX Context objects
rtDeclareVariable(Ray, ray, rtCurrentRay, );                 // current ray
rtDeclareVariable(PerRayData, prd, rtPayload, );             // ray PRD
rtDeclareVariable(HitRecord, hit_rec, attribute hit_rec, );  // from geometry

// Material Parameters
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3, int)>,
                  sample_texture, , );
rtDeclareVariable(float, rA, , );
rtDeclareVariable(float, rB, , );

///////////////////////////
// --- BRDF Programs --- //
///////////////////////////

// TODO: idea, the closest hit can get the surface parameters as normal and then
// call a templated function, that then calls the sample and other functions as
// normal

// Assigns material and hit parameters to PRD
RT_PROGRAM void closest_hit() {
  prd.matType = Oren_Nayar_BRDF;
  prd.isSpecular = false;
  prd.scatterEvent = rayGotBounced;

  // Get hit params
  prd.origin = hit_rec.p;
  prd.geometric_normal = normalize(hit_rec.geometric_normal);
  prd.shading_normal = normalize(hit_rec.shading_normal);
  prd.view_direction = normalize(hit_rec.view_direction);

  // Get material color
  int index = hit_rec.index;
  float3 color = sample_texture(hit_rec.u, hit_rec.v, hit_rec.p, index);

  // Assign material parameters to PRD, to be used in the sampling programs
  prd.matParams.orenNayar.rA = rA;
  prd.matParams.orenNayar.rB = rB;
  prd.matParams.attenuation = color;
}

RT_CALLABLE_PROGRAM float3 BRDF_Sample(const BRDFParameters &surface,
                                       const float3 &P,   // next ray origin
                                       const float3 &Wo,  // prev ray direction
                                       const float3 &N,   // shading normal
                                       uint &seed) {
  float3 Wi;
  cosine_sample_hemisphere(rnd(seed), rnd(seed), Wi);

  Onb uvw(N);
  uvw.inverse_transform(Wi);

  return Wi;
}

RT_CALLABLE_PROGRAM float BRDF_PDF(const BRDFParameters &surface,
                                   const float3 &P,    // next ray origin
                                   const float3 &Wo,   // prev ray direction
                                   const float3 &Wi,   // next ray direction
                                   const float3 &N) {  // shading normal
  float cosine = dot(normalize(Wi), normalize(N));

  if (cosine < 0.f)
    return 0.f;
  else
    return cosine / PI_F;
}

// Evaluates BRDF, returning its reflectance
RT_CALLABLE_PROGRAM float3
BRDF_Evaluate(const BRDFParameters &surface,
              const float3 &P,    // next ray origin
              const float3 &Wo,   // prev ray direction
              const float3 &Wi,   // next ray direction
              const float3 &N) {  // shading normal
  float3 WiN = normalize(Wi);

  float sinThetaI = SinTheta(WiN);
  float sinThetaO = SinTheta(Wo);
  // Compute cosine term of Oren-Nayar model
  float maxCos = 0;
  if (sinThetaI > 1e-4 && sinThetaO > 1e-4) {
    float sinPhiI = SinPhi(WiN), cosPhiI = CosPhi(WiN);
    float sinPhiO = SinPhi(Wo), cosPhiO = CosPhi(Wo);
    float dCos = cosPhiI * cosPhiO + sinPhiI * sinPhiO;
    maxCos = fmaxf(0.f, dCos);
  }

  // Compute sine and tangent terms of Oren-Nayar model
  float sinAlpha, tanBeta;
  if (AbsCosTheta(WiN) > AbsCosTheta(Wo)) {
    sinAlpha = sinThetaO;
    tanBeta = sinThetaI / AbsCosTheta(WiN);
  } else {
    sinAlpha = sinThetaI;
    tanBeta = sinThetaO / AbsCosTheta(Wo);
  }

  float rA = surface.orenNayar.rA;
  float rB = surface.orenNayar.rB;
  float3 color = surface.attenuation;

  return color * (1.f / PI_F) * (rA + rB * maxCos * sinAlpha * tanBeta);
}