#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "material.cuh"

/*! the implicit state's ray we will intersect against */
rtDeclareVariable(Ray, ray, rtCurrentRay, );

/*! the per ray data we operate on */
rtDeclareVariable(PerRayData, prd, rtPayload, );
rtDeclareVariable(rtObject, world, , );

/*! the attributes we use to communicate between intersection programs and hit
 * program */
rtDeclareVariable(HitRecord, hit_rec, attribute hit_rec, );

/*! and finally - that particular material's parameters */
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3, int)>,
                  sample_texture, , );

// TODO: add geometric_normal and shading_normal params to PRD

RT_PROGRAM void closest_hit() {
  // get material params from buffer
  int texIndex = hit_rec.index;

  // assign material params to prd
  prd.matType = Lambertian_Material;
  prd.isSpecular = false;
  prd.scatterEvent = rayGotBounced;

  prd.emitted = make_float3(0.f);
  prd.attenuation = sample_texture(hit_rec.u, hit_rec.v, hit_rec.p, texIndex);

  // assign hit params to prd
  prd.origin = hit_rec.p;
  prd.geometric_normal = hit_rec.geometric_normal;
  prd.shading_normal = hit_rec.shading_normal;
}

RT_CALLABLE_PROGRAM float3 BRDF_Sample(PDFParams &pdf, uint &seed) {
  float3 temp;
  cosine_sample_hemisphere(rnd(seed), rnd(seed), temp);

  Onb uvw(pdf.normal);
  uvw.inverse_transform(temp);

  pdf.direction = temp;

  return pdf.direction;
}

RT_CALLABLE_PROGRAM float BRDF_PDF(PDFParams &pdf) {
  float cosine = dot(unit_vector(pdf.direction), unit_vector(pdf.normal));

  if (cosine < 0.f)
    return 0.f;
  else
    return cosine / PI_F;
}

RT_CALLABLE_PROGRAM float BRDF_Evaluate(PDFParams &pdf) {
  float cosine = dot(unit_vector(pdf.direction), unit_vector(pdf.normal));

  if (cosine < 0.f)
    return 0.f;
  else
    return cosine / PI_F;
}