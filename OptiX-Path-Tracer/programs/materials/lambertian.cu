#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "light_sample.cuh"

///////////////////////////////////
// --- Lambertian BRDF Model --- //
///////////////////////////////////

// OptiX Context objects
rtDeclareVariable(Ray, ray, rtCurrentRay, );                 // current ray
rtDeclareVariable(PerRayData, prd, rtPayload, );             // ray PRD
rtDeclareVariable(rtObject, world, , );                      // scene graph
rtDeclareVariable(HitRecord, hit_rec, attribute hit_rec, );  // from geometry

// Material Parameters
rtDeclareVariable(Texture_Function, sample_texture, , );

RT_FUNCTION Lambertian_Parameters Get_Parameters(const float3 &P, float u,
                                                 float v, int index) {
  Lambertian_Parameters surface;

  surface.color = sample_texture(u, v, P, index);

  return surface;
}

// Lambertian Material Closest Hit Program
RT_PROGRAM void closest_hit() {
  int index = hit_rec.index;
  float u = hit_rec.u, v = hit_rec.v;
  float3 P = hit_rec.p, Wo = hit_rec.view_direction;
  float3 N = hit_rec.shading_normal;

  Lambertian_Parameters surface = Get_Parameters(P, u, v, index);

  // Sample Direct Light
  float3 direct = Direct_Light(surface, P, Wo, N, false, prd.seed);
  prd.radiance += prd.throughput * direct;

  // Sample BRDF
  float3 Wi = Sample(surface, P, Wo, N, prd.seed);
  float pdf;  // calculated in the Evaluate function
  float3 attenuation = Evaluate(surface, P, Wo, Wi, N, pdf);

  // Assign parameters to PRD
  prd.scatterEvent = rayGotBounced;
  prd.origin = hit_rec.p;
  prd.direction = Wi;
  prd.throughput *= clamp(attenuation / pdf, 0.f, 1.f);
  prd.isSpecular = false;
}