#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "light_sample.cuh"

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData, prd, rtPayload, );
rtDeclareVariable(rtObject, world, , );
rtDeclareVariable(HitRecord, hit_rec, attribute hit_rec, );

/*! and finally - that particular material's parameters */
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3, int)>,
                  sample_texture, , );

RT_FUNCTION Lambertian_Parameters Get_Parameters(const float3 &P, float u,
                                                 float v, int index) {
  Lambertian_Parameters surface;

  surface.color = sample_texture(u, v, P, index);

  return surface;
}

// Lambertian Material Closest Hit Program
RT_PROGRAM void closest_hit() {
  int index = hit_rec.index;
  float u = hit_rec.u, v = hit_rec.v;
  float3 P = hit_rec.p, Wo = hit_rec.view_direction;
  float3 N = hit_rec.shading_normal;

  Lambertian_Parameters surface = Get_Parameters(P, u, v, index);

  // Sample Direct Light
  float3 direct = Direct_Light(surface, P, Wo, N, false, prd.seed);
  prd.radiance += prd.throughput * direct;

  // Sample BRDF
  float3 Wi = Sample(surface, P, Wo, N, prd.seed);
  float pdf;  // calculated in the Evaluate function
  float3 attenuation = Evaluate(surface, P, Wo, Wi, N, pdf);

  // Assign parameters to PRD
  prd.scatterEvent = rayGotBounced;
  prd.origin = hit_rec.p;
  prd.direction = Wi;
  prd.throughput *= clamp(attenuation / pdf, 0.f, 1.f);
}