#include "hip/hip_runtime.h"
#include "light_sample.cuh"

////////////////////////////////////////////////////////////
// --- Ashikhmin-Shirley Anisotropic Phong BRDF Model --- //
////////////////////////////////////////////////////////////

// Original Paper & Tech Report - "An Anisotropic Phong Light Reflection Model"
// https://www.cs.utah.edu/~shirley/papers/jgtbrdf.pdf
// https://www.cs.utah.edu/docs/techreports/2000/pdf/UUCS-00-014.pdf

// Reference Implementation:
// https://developer.blender.org/diffusion/C/browse/master/src/kernel/closure/bsdf_ashikhmin_shirley.h
// FresnelBlend from PBRT
// https://github.com/mmp/pbrt-v3/blob/9f717d847a807793fa966cf0eaa366852efef167/src/core/reflection.cpp
// https://github.com/mmp/pbrt-v3/blob/9f717d847a807793fa966cf0eaa366852efef167/src/core/reflection.h

// OptiX Context objects
rtDeclareVariable(Ray, ray, rtCurrentRay, );                 // current ray
rtDeclareVariable(PerRayData, prd, rtPayload, );             // ray PRD
rtDeclareVariable(rtObject, world, , );                      // scene graph
rtDeclareVariable(HitRecord, hit_rec, attribute hit_rec, );  // from geometry

// Material Parameters
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3, int)>,
                  diffuse_color, , );
rtDeclareVariable(rtCallableProgramId<float3(float, float, float3, int)>,
                  specular_color, , );
rtDeclareVariable(float, nu, , );
rtDeclareVariable(float, nv, , );

RT_FUNCTION Ashikhmin_Shirley_Parameters Get_Parameters(const float3 &P,
                                                        float u, float v,
                                                        int index) {
  Ashikhmin_Shirley_Parameters surface;

  surface.diffuse_color = diffuse_color(u, v, P, index);
  surface.specular_color = specular_color(u, v, P, index);
  surface.nu = nu;
  surface.nv = nv;

  return surface;
}

// Assigns material and hit parameters to PRD
RT_PROGRAM void closest_hit() {
  int index = hit_rec.index;
  float u = hit_rec.u, v = hit_rec.v;
  float3 P = hit_rec.p, Wo = hit_rec.view_direction;
  float3 N = hit_rec.shading_normal;

  Ashikhmin_Shirley_Parameters surface = Get_Parameters(P, u, v, index);

  // Sample BRDF
  float3 Wi = Sample(surface, P, Wo, N, prd.seed);
  float pdf;  // calculated in the Evaluate function
  float3 attenuation = Evaluate(surface, P, Wo, Wi, N, pdf);

  // Assign parameters to PRD
  prd.scatterEvent = rayGotBounced;
  prd.origin = hit_rec.p;
  prd.direction = Wi;
  prd.throughput *= clamp(attenuation, 0.f, 1.f);
  prd.isSpecular = true;
}