#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "material.h"

// the implicit state's ray we will intersect against
rtDeclareVariable(Ray, ray, rtCurrentRay, );

// the per ray data we operate on
rtDeclareVariable(PerRayData, prd, rtPayload, );
rtDeclareVariable(rtObject, world, , );

// the attributes we use to communicate between intersection programs and hit
// program
rtDeclareVariable(HitRecord, hit_rec, attribute hit_rec, );

// and finally - that particular material's parameters
rtBuffer<rtCallableProgramId<float3(float, float, float3)> > sample_texture;
rtDeclareVariable(float, fuzz, , );  // how 'rough'/fuzzy the metal is

RT_PROGRAM void closest_hit() {
  prd.matType = Metal_Material;
  prd.isSpecular = true;
  prd.scatterEvent = rayGotBounced;

  prd.origin = hit_rec.p;
  prd.normal = hit_rec.normal;

  float3 reflected = reflect(unit_vector(ray.direction), prd.normal);
  prd.direction = reflected + fuzz * random_in_unit_sphere(prd.seed);

  int index = hit_rec.index;
  prd.emitted = make_float3(0.f);
  prd.attenuation = sample_texture[index](hit_rec.u, hit_rec.v, hit_rec.p);
}

RT_CALLABLE_PROGRAM float3 BRDF_Sample(PDFParams &pdf, uint &seed) {
  return make_float3(1.f);
}

RT_CALLABLE_PROGRAM float BRDF_PDF(PDFParams &pdf) { return 1.f; }

RT_CALLABLE_PROGRAM float BRDF_Evaluate(PDFParams &pdf) { return 1.f; }
