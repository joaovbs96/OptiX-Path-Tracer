#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "material.h"

// the implicit state's ray we will intersect against
rtDeclareVariable(Ray, ray, rtCurrentRay, );

// the per ray data we operate on
rtDeclareVariable(PerRayData, prd, rtPayload, );
rtDeclareVariable(rtObject, world, , );

// the attributes we use to communicate between intersection programs and hit
// program
rtDeclareVariable(Hit_Record, hit_rec, attribute hit_rec, );

// and finally - that particular material's parameters
rtBuffer<rtCallableProgramId<float3(float, float, float3)> > sample_texture;
rtDeclareVariable(float, fuzz, , );

RT_FUNCTION bool scatter(const Ray &ray_in) {
  float3 reflected = reflect(unit_vector(ray_in.direction), hit_rec.normal);
  prd.out.is_specular = true;
  prd.out.origin = hit_rec.p;
  prd.out.direction =
      reflected + fuzz * random_in_unit_sphere((*prd.in.randState));
  prd.out.attenuation =
      sample_texture[hit_rec.index](hit_rec.u, hit_rec.v, hit_rec.p);
  prd.out.normal = hit_rec.normal;
  return true;
}

RT_FUNCTION float3 emitted() { return make_float3(0.f, 0.f, 0.f); }

RT_PROGRAM void closest_hit() {
  prd.out.type = Metal;
  prd.out.emitted = emitted();
  prd.out.scatterEvent = scatter(ray) ? rayGotBounced : rayGotCancelled;
}
