#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "pdfs/pdf.cuh"
#include "prd.cuh"

// launch index and frame dimensions
rtDeclareVariable(uint2, pixelID, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

// ray related state
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData, prd, rtPayload, );

rtBuffer<float4, 2> acc_buffer;      // HDR color frame buffer
rtBuffer<uchar4, 2> display_buffer;  // display buffer

rtDeclareVariable(int, samples, , );   // number of samples
rtDeclareVariable(int, frame, , );     // frame number
rtDeclareVariable(int, pixelDim, , );  // pxiel dimension for stratification

rtDeclareVariable(rtObject, world, , );  // scene/top obj variable

// Camera parameters
rtDeclareVariable(float3, camera_lower_left_corner, , );
rtDeclareVariable(float3, camera_horizontal, , );
rtDeclareVariable(float3, camera_vertical, , );
rtDeclareVariable(float3, camera_origin, , );
rtDeclareVariable(float3, camera_u, , );
rtDeclareVariable(float3, camera_v, , );
rtDeclareVariable(float, camera_lens_radius, , );
rtDeclareVariable(float, time0, , );
rtDeclareVariable(float, time1, , );

// Light sampling callable programs
rtDeclareVariable(int, numLights, , );
rtBuffer<float3> Light_Emissions;
rtBuffer<rtCallableProgramId<float3(PDFParams&, uint&)>> Light_Sample;
rtBuffer<rtCallableProgramId<float(PDFParams&)>> Light_PDF;

// BRDF sampling callable programs
rtBuffer<rtCallableProgramId<float3(PDFParams&, uint&)>> BRDF_Sample;
rtBuffer<rtCallableProgramId<float(PDFParams&)>> BRDF_PDF;
rtBuffer<rtCallableProgramId<float3(PDFParams&)>> BRDF_Evaluate;

RT_FUNCTION float PowerHeuristic(unsigned int numf, float fPdf,
                                 unsigned int numg, float gPdf) {
  float f = numf * fPdf;
  float g = numg * gPdf;

  return (f * f) / (f * f + g * g);
}

RT_FUNCTION float3 Direct_Light(PerRayData& prd) {
  float3 directLight = make_float3(0.f);

  // return black if there's no light
  if (numLights == 0) return make_float3(0.f);

  // return black if ray missed
  if (prd.scatterEvent == rayMissed) return make_float3(0.f);

  // ramdomly pick one light and multiply the result by the number of lights
  // it's the same as dividing by the PDF if they have the same probability
  int index = ((int)(rnd(prd.seed) * numLights)) % numLights;

  // return black if there's just one light and we just hit it
  if (prd.matType == Diffuse_Light_BRDF) {
    if (numLights == 1) return make_float3(0.f);
  }

  // Sample Light
  float3 emission = Light_Emissions[index];
  PDFParams pdfParams(prd);
  Light_Sample[index](pdfParams, prd.seed);
  float lightPDF = Light_PDF[index](pdfParams);

  // only sample if surface normal is in the light direction
  if (dot(pdfParams.direction, pdfParams.normal) < 0.f) return make_float3(0.f);

  // Check if light is occluded
  PerRayData_Shadow prdShadow;
  Ray shadowRay = make_Ray(/* origin   : */ pdfParams.origin,
                           /* direction: */ pdfParams.direction,
                           /* ray type : */ 1,
                           /* tmin     : */ 1e-3f,
                           /* tmax     : */ RT_DEFAULT_MAX);
  rtTrace(world, shadowRay, prdShadow);

  // if light is occluded, return black
  if (prdShadow.inShadow) return make_float3(0.f);

  // Multiple Importance Sample

  // Sample light
  if (lightPDF != 0.f && !isNull(emission)) {
    float matPDF = BRDF_PDF[prd.matType](pdfParams);
    float3 matValue = BRDF_Evaluate[prd.matType](pdfParams);
    if (matPDF != 0.f && !isNull(matValue)) {
      float weight = PowerHeuristic(1, lightPDF, 1, matPDF);
      directLight += matValue * emission * weight / lightPDF;
    }
  }

  // Sample BRDF
  BRDF_Sample[prd.matType](pdfParams, prd.seed);
  float matPDF = BRDF_PDF[prd.matType](pdfParams);
  float3 matValue = BRDF_Evaluate[prd.matType](pdfParams);
  if (matPDF != 0.f && !isNull(matValue)) {
    lightPDF = Light_PDF[index](pdfParams);

    // we didn't hit anything, ignore BRDF sample
    if (!lightPDF || isNull(emission)) return directLight;

    float weight = PowerHeuristic(1, matPDF, 1, lightPDF);
    directLight += matValue * emission * weight / matPDF;
  }

  return directLight;
}

struct Camera {
  static RT_FUNCTION Ray generateRay(float s, float t, uint& seed) {
    const float3 rd = camera_lens_radius * random_in_unit_disk(seed);
    const float3 lens_offset = camera_u * rd.x + camera_v * rd.y;
    const float3 origin = camera_origin + lens_offset;
    const float3 direction = camera_lower_left_corner + s * camera_horizontal +
                             t * camera_vertical - origin;

    return make_Ray(/* origin   : */ origin,
                    /* direction: */ direction,
                    /* ray type : */ 0,
                    /* tmin     : */ 1e-6f,
                    /* tmax     : */ RT_DEFAULT_MAX);
  }
};

// Check if we should take emissions into account, in the next light hit
RT_FUNCTION bool Emission_Next(BRDFType type) {
  switch (type) {
    case Metal_BRDF:
    case Dielectric_BRDF:
    case Isotropic_BRDF:
    case Anisotropic_BRDF:
    case Torrance_Sparrow_BRDF:
      return true;
    default:
      return false;
  }
}

// Check if current BRDF should directly sample light
RT_FUNCTION bool Do_Direct_Sampling(BRDFType type) {
  switch (type) {
    case Metal_BRDF:
    case Dielectric_BRDF:
    case Anisotropic_BRDF:
    case Torrance_Sparrow_BRDF:
      return false;
    default:
      return true;
  }
}

RT_FUNCTION float3 color(Ray& ray, uint& seed) {
  PerRayData prd;
  prd.seed = seed;
  prd.time = time0 + rnd(prd.seed) * (time1 - time0);

  prd.throughput = make_float3(1.f);
  float3 radiance = make_float3(0.f);
  bool previousHitSpecular = false;

  // TODO: Isotropic isn't working when light is present

  // iterative version of recursion
  for (int depth = 0; depth < 50; depth++) {
    rtTrace(world, ray, prd);  // Trace a new ray

    // if the material is the normal shader, return its color
    if (prd.matType == Normal_BRDF) return prd.attenuation;

    // Only sample direct light if last bounce wasn't specular
    if (Do_Direct_Sampling(prd.matType))
      radiance += prd.throughput * Direct_Light(prd);

    // ray got 'lost' to the environment
    // return attenuation set by miss shader
    if (prd.scatterEvent == rayMissed) {
      radiance += prd.throughput * prd.attenuation;
      return radiance;
    }

    // ray hit a light, return radiance
    else if (prd.scatterEvent == rayGotCancelled) {
      // Take care not to double dip
      if (depth == 0 || previousHitSpecular)
        radiance += prd.throughput * prd.emitted;

      return radiance;
    }

    // ray is still alive, and got properly bounced
    else {
      // if it was an ideal specular hit, accumulate color
      if (prd.isSpecular) prd.throughput *= prd.attenuation;

      // otherwise, do importance sample
      else {
        // Sample BRDF
        // TODO: use PRD
        PDFParams pdfParams(prd);
        BRDF_Sample[prd.matType](pdfParams, seed);
        float matPDF = BRDF_PDF[prd.matType](pdfParams);
        float3 matValue = BRDF_Evaluate[prd.matType](pdfParams);

        if (matPDF == 0.f) return make_float3(0.f);

        // Accumulate color
        prd.throughput *= clamp(matValue / matPDF, 0.f, 1.f);

        // Update ray origin and direction
        prd.origin = pdfParams.origin;
        prd.direction = pdfParams.direction;
      }

      // check if emissions should be taken into account if we hit a light
      previousHitSpecular = Emission_Next(prd.matType);

      // generate a new ray
      ray = make_Ray(/* origin   : */ prd.origin,
                     /* direction: */ prd.direction,
                     /* ray type : */ 0,
                     /* tmin     : */ 1e-3f,
                     /* tmax     : */ RT_DEFAULT_MAX);
    }

    // Russian Roulette Path Termination
    float p = max_component(prd.throughput);
    if (depth > 10) {
      if (rnd(prd.seed) >= p)
        return radiance + prd.throughput * prd.attenuation;
      else
        prd.throughput *= 1.f / p;
    }
  }

  // recursion did not terminate - cancel it
  return make_float3(0.f);
}

// Remove NaN values
RT_FUNCTION float3 de_nan(const float3& c) {
  float3 temp = c;

  if (!(temp.x == temp.x)) temp.x = 0.f;
  if (!(temp.y == temp.y)) temp.y = 0.f;
  if (!(temp.z == temp.z)) temp.z = 0.f;

  return temp;
}

RT_FUNCTION uchar4 make_Color(float4 col) {
  float3 temp = sqrt(make_float3(col.x, col.y, col.z) / (frame + 1));
  temp = clamp(temp, 0.f, 1.f);

  int r = int(255.99 * temp.x);  // R
  int g = int(255.99 * temp.y);  // G
  int b = int(255.99 * temp.z);  // B
  int a = int(255.99 * 1.f);     // A

  return make_uchar4(r, g, b, a);
}

RT_PROGRAM void renderPixel() {
  // get RNG seed
  uint seed = tea<64>(launchDim.x * pixelID.y + pixelID.x, frame);

  // initialize acc buffer if needed
  uint2 index = make_uint2(pixelID.x, launchDim.y - pixelID.y - 1);
  if (frame == 0) acc_buffer[index] = make_float4(0.f);

  float3 col = make_float3(0.f);
  for (int i = 0; i < pixelDim; i++) {
    for (int j = 0; j < pixelDim; j++) {
      // Subpixel jitter: send the ray through a different position inside the
      // pixel each time, to provide antialiasing.
      float u = float(pixelID.x + (i + rnd(seed)) / pixelDim) / launchDim.x;
      float v = float(pixelID.y + (j + rnd(seed)) / pixelDim) / launchDim.y;

      // trace ray
      Ray ray = Camera::generateRay(u, v, seed);

      // accumulate subpixel color
      col += de_nan(color(ray, seed));
    }
  }

  // average subpixel sum
  col /= (pixelDim * pixelDim);

  // accumulate pixel color
  acc_buffer[index] += make_float4(col.x, col.y, col.z, 1.f);
  display_buffer[index] = make_Color(acc_buffer[index]);
}