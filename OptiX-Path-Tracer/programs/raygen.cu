#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "pdfs/pdf.h"
#include "prd.h"

// the 'builtin' launch index we need to render a frame
rtDeclareVariable(uint2, pixelID, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

// the ray related state
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData, prd, rtPayload, );

rtBuffer<float3, 2> fb;          // float3 color frame buffer
rtBuffer<unsigned int, 2> seed;  // uint seed buffer

rtDeclareVariable(int, samples, , );  // number of samples
rtDeclareVariable(int, frame, , );    // current frame

rtDeclareVariable(rtObject, world, , );  // scene variable

rtDeclareVariable(float3, camera_lower_left_corner, , );
rtDeclareVariable(float3, camera_horizontal, , );
rtDeclareVariable(float3, camera_vertical, , );
rtDeclareVariable(float3, camera_origin, , );
rtDeclareVariable(float3, camera_u, , );
rtDeclareVariable(float3, camera_v, , );
rtDeclareVariable(float, camera_lens_radius, , );
rtDeclareVariable(float, time0, , );
rtDeclareVariable(float, time1, , );

// Light sampling callable programs
rtDeclareVariable(int, numLights, , );
rtBuffer<float3> Light_Emissions;
rtBuffer<rtCallableProgramId<float3(PDFParams&, XorShift32&)>> Light_Sample;
rtBuffer<rtCallableProgramId<float(PDFParams&)>> Light_PDF;

// BRDF sampling callable programs
rtBuffer<rtCallableProgramId<float3(PDFParams&, XorShift32&)>> BRDF_Sample;
rtBuffer<rtCallableProgramId<float(PDFParams&)>> BRDF_PDF;
rtBuffer<rtCallableProgramId<float(PDFParams&)>> BRDF_Evaluate;

RT_FUNCTION float PowerHeuristic(unsigned int numf, float fPdf,
                                 unsigned int numg, float gPdf) {
  float f = numf * fPdf;
  float g = numg * gPdf;

  return (f * f) / (f * f + g * g);
}

RT_FUNCTION float3 Direct_Light(PerRayData& prd) {
  // return black if there's no light
  if (numLights == 0) return make_float3(0.f);

  // cancel if it's a specular bounce
  if (prd.isSpecular) return make_float3(0.f);

  // ramdomly pick one light and multiply the result by the number of lights
  // it's the same as dividing by the PDF if they have the same probability
  int index = ((int)((*prd.randState)() * numLights)) % numLights;

  // return black if there's just one light and we just hit it
  if (prd.matType == Diffuse_Light_Material) {
    if (numLights == 1) return make_float3(0.f);
  }

  PDFParams pdfParams(prd.origin, prd.normal);
  Light_Sample[index](pdfParams, *prd.randState);
  float lightPDF = Light_PDF[index](pdfParams);

  if (dot(pdfParams.direction, pdfParams.normal) <= 0.f)
    return make_float3(0.f);

  PerRayData_Shadow prdShadow;
  Ray shadowRay = make_Ray(/* origin   : */ pdfParams.origin,
                           /* direction: */ pdfParams.direction,
                           /* ray type : */ 1,
                           /* tmin     : */ 1e-3f,
                           /* tmax     : */ RT_DEFAULT_MAX);
  rtTrace(world, shadowRay, prdShadow);

  // if light is occluded, return black
  if (prdShadow.inShadow) return make_float3(0.f);

  float matPDF = BRDF_PDF[prd.matType](pdfParams);
  float3 matValue = prd.attenuation * BRDF_Evaluate[prd.matType](pdfParams);

  // MIS
  float3 emission = Light_Emissions[index];
  float3 lightThroughput = matValue * prd.throughput * numLights * emission;
  lightThroughput *= PowerHeuristic(1, lightPDF, 1, matPDF);
  lightThroughput /= max(0.001f, lightPDF);

  return lightThroughput;
}

struct Camera {
  static RT_FUNCTION Ray generateRay(float s, float t, XorShift32& rnd) {
    const float3 rd = camera_lens_radius * random_in_unit_disk(rnd);
    const float3 lens_offset = camera_u * rd.x + camera_v * rd.y;
    const float3 origin = camera_origin + lens_offset;
    const float3 direction = camera_lower_left_corner + s * camera_horizontal +
                             t * camera_vertical - origin;

    return make_Ray(/* origin   : */ origin,
                    /* direction: */ direction,
                    /* ray type : */ 0,
                    /* tmin     : */ 1e-6f,
                    /* tmax     : */ RT_DEFAULT_MAX);
  }
};

RT_FUNCTION float3 color(Ray& ray, XorShift32& rnd) {
  PerRayData prd;
  prd.randState = &rnd;
  prd.time = time0 + rnd() * (time1 - time0);

  prd.throughput = make_float3(1.f);
  float3 radiance = make_float3(0.f);

  // iterative version of recursion, up to depth 50
  for (int depth = 0; depth < 50; depth++) {
    rtTrace(world, ray, prd);

    radiance += Direct_Light(prd) * prd.throughput;

    // ray got 'lost' to the environment
    // return attenuation set by miss shader
    if (prd.scatterEvent == rayMissed) {
      radiance += prd.throughput * prd.attenuation;
      break;
    }

    // ray hit a light, return emission
    else if (prd.scatterEvent == rayGotCancelled) {
      radiance += prd.throughput * prd.emitted;
      break;
    }

    // ray is still alive, and got properly bounced
    else {
      // ideal specular hit
      if (prd.isSpecular) prd.throughput *= prd.attenuation;

      // do importance sample
      else {
        PDFParams pdfParams(prd.origin, prd.normal);
        BRDF_Sample[prd.matType](pdfParams, rnd);
        float pdfValue = BRDF_PDF[prd.matType](pdfParams);

        prd.attenuation *= BRDF_Evaluate[prd.matType](pdfParams);

        prd.throughput *= prd.attenuation / pdfValue;

        prd.origin = pdfParams.origin;
        prd.direction = pdfParams.direction;
      }

      ray = make_Ray(/* origin   : */ prd.origin,
                     /* direction: */ prd.direction,
                     /* ray type : */ 0,
                     /* tmin     : */ 1e-3f,
                     /* tmax     : */ RT_DEFAULT_MAX);
    }

    // Russian Roulette Path Termination
    float p = max_component(prd.throughput);
    if (depth > 10) {
      if (rnd() >= p)
        return prd.throughput;
      else
        prd.throughput *= 1.f / p;
    }
  }

  // recursion did not terminate - cancel it
  return radiance;
}

// Remove NaN values
RT_FUNCTION float3 de_nan(const float3& c) {
  float3 temp = c;
  if (!(temp.x == temp.x)) temp.x = 0.f;
  if (!(temp.y == temp.y)) temp.y = 0.f;
  if (!(temp.z == temp.z)) temp.z = 0.f;

  return temp;
}

/*! the actual ray generation program - note this has no formal
  function parameters, but gets its paramters throught the 'pixelID'
  and 'pixelBuffer' variables/buffers declared above */
RT_PROGRAM void renderPixel() {
  XorShift32 rnd;

  // init frame buffer and rng
  if (frame == 0) {
    unsigned int init_index = pixelID.y * launchDim.x + pixelID.x;
    rnd.init(init_index);

    // initiate the color buffer
    fb[pixelID] = make_float3(0.f, 0.f, 0.f);
  } else
    rnd.state = seed[pixelID];

  // Subpixel jitter: send the ray through a different position inside the pixel
  // each time, to provide antialiasing.
  float u = float(pixelID.x + rnd()) / float(launchDim.x);
  float v = float(pixelID.y + rnd()) / float(launchDim.y);

  // trace ray
  Ray ray = Camera::generateRay(u, v, rnd);

  fb[pixelID] += de_nan(color(ray, rnd));  // accumulate color
  seed[pixelID] = rnd.state;               // save RND state
}
