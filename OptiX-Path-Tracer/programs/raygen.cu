#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "pdfs/pdf.cuh"
#include "prd.cuh"

// launch index and frame dimensions
rtDeclareVariable(uint2, pixelID, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

// ray related state
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData, prd, rtPayload, );

rtBuffer<float4, 2> acc_buffer;      // HDR color frame buffer
rtBuffer<uchar4, 2> display_buffer;  // display buffer

rtDeclareVariable(int, samples, , );  // number of samples
rtDeclareVariable(int, frame, , );    // frame number

rtDeclareVariable(rtObject, world, , );  // scene/top obj variable

// Camera parameters
rtDeclareVariable(float3, camera_lower_left_corner, , );
rtDeclareVariable(float3, camera_horizontal, , );
rtDeclareVariable(float3, camera_vertical, , );
rtDeclareVariable(float3, camera_origin, , );
rtDeclareVariable(float3, camera_u, , );
rtDeclareVariable(float3, camera_v, , );
rtDeclareVariable(float, camera_lens_radius, , );
rtDeclareVariable(float, time0, , );
rtDeclareVariable(float, time1, , );

// Light sampling callable programs
rtDeclareVariable(int, numLights, , );
rtBuffer<float3> Light_Emissions;
rtBuffer<rtCallableProgramId<float3(PDFParams&, uint&)>> Light_Sample;
rtBuffer<rtCallableProgramId<float(PDFParams&)>> Light_PDF;

// BRDF sampling callable programs
rtBuffer<rtCallableProgramId<float3(PDFParams&, uint&)>> BRDF_Sample;
rtBuffer<rtCallableProgramId<float(PDFParams&)>> BRDF_PDF;
rtBuffer<rtCallableProgramId<float(PDFParams&)>> BRDF_Evaluate;

RT_FUNCTION float PowerHeuristic(unsigned int numf, float fPdf,
                                 unsigned int numg, float gPdf) {
  float f = numf * fPdf;
  float g = numg * gPdf;

  return (f * f) / (f * f + g * g);
}

RT_FUNCTION float3 Direct_Light(PerRayData& prd) {
  // return black if there's no light
  if (numLights == 0) return make_float3(0.f);

  // ramdomly pick one light and multiply the result by the number of lights
  // it's the same as dividing by the PDF if they have the same probability
  int index = ((int)(rnd(prd.seed) * numLights)) % numLights;

  // return black if there's just one light and we just hit it
  if (prd.matType == Diffuse_Light_Material) {
    if (numLights == 1) return make_float3(0.f);
  }

  // Sample Light
  PDFParams pdfParams(prd.origin, prd.normal);
  Light_Sample[index](pdfParams, prd.seed);
  float lightPDF = Light_PDF[index](pdfParams);
  // TODO: check if pdf value is 0

  if (dot(pdfParams.direction, pdfParams.normal) <= 0.f)
    return make_float3(0.f);

  // Check if light is occluded
  PerRayData_Shadow prdShadow;
  Ray shadowRay = make_Ray(/* origin   : */ pdfParams.origin,
                           /* direction: */ pdfParams.direction,
                           /* ray type : */ 1,
                           /* tmin     : */ 1e-3f,
                           /* tmax     : */ RT_DEFAULT_MAX);
  rtTrace(world, shadowRay, prdShadow);

  // if light is occluded, return black
  if (prdShadow.inShadow) return make_float3(0.f);

  // Sample BRDF
  float matPDF = BRDF_PDF[prd.matType](pdfParams);
  // TODO: check if pdf value is 0
  float3 matValue = prd.attenuation * BRDF_Evaluate[prd.matType](pdfParams);

  // MIS
  float3 emission = Light_Emissions[index];
  float3 lightThroughput = matValue * prd.throughput * numLights * emission;
  lightThroughput *= PowerHeuristic(1, lightPDF, 1, matPDF);
  lightThroughput /= max(0.001f, lightPDF);

  return lightThroughput;
}

struct Camera {
  static RT_FUNCTION Ray generateRay(float s, float t, uint& seed) {
    const float3 rd = camera_lens_radius * random_in_unit_disk(seed);
    const float3 lens_offset = camera_u * rd.x + camera_v * rd.y;
    const float3 origin = camera_origin + lens_offset;
    const float3 direction = camera_lower_left_corner + s * camera_horizontal +
                             t * camera_vertical - origin;

    return make_Ray(/* origin   : */ origin,
                    /* direction: */ direction,
                    /* ray type : */ 0,
                    /* tmin     : */ 1e-6f,
                    /* tmax     : */ RT_DEFAULT_MAX);
  }
};

RT_FUNCTION float3 color(Ray& ray, uint& seed) {
  PerRayData prd;
  prd.seed = seed;
  prd.time = time0 + rnd(prd.seed) * (time1 - time0);

  prd.throughput = make_float3(1.f);
  float3 radiance = make_float3(0.f);

  // iterative version of recursion, up to depth 50
  for (int depth = 0; depth < 50; depth++) {
    rtTrace(world, ray, prd);

    // Only sample direct light if last bounce wasn't specular
    if (!prd.isSpecular) radiance += prd.throughput * Direct_Light(prd);

    // ray got 'lost' to the environment
    // return attenuation set by miss shader
    if (prd.scatterEvent == rayMissed) {
      radiance += prd.throughput * prd.attenuation;
      break;
    }

    // ray hit a light, return emission
    else if (prd.scatterEvent == rayGotCancelled) {
      radiance += prd.throughput * prd.emitted;
      break;
    }

    // ray is still alive, and got properly bounced
    else {
      // ideal specular hit
      if (prd.isSpecular) prd.throughput *= prd.attenuation;

      // do importance sample
      else {
        PDFParams pdfParams(prd.origin, prd.normal);
        BRDF_Sample[prd.matType](pdfParams, seed);
        float pdfValue = BRDF_PDF[prd.matType](pdfParams);
        // TODO: check if pdf value is 0

        prd.attenuation *= BRDF_Evaluate[prd.matType](pdfParams);

        prd.throughput *= prd.attenuation / pdfValue;
        prd.throughput = Clamp(prd.throughput);

        prd.origin = pdfParams.origin;
        prd.direction = pdfParams.direction;
      }

      ray = make_Ray(/* origin   : */ prd.origin,
                     /* direction: */ prd.direction,
                     /* ray type : */ 0,
                     /* tmin     : */ 1e-3f,
                     /* tmax     : */ RT_DEFAULT_MAX);
    }

    // Russian Roulette Path Termination
    float p = max_component(prd.throughput);
    if (depth > 10) {
      if (rnd(prd.seed) >= p)
        return prd.throughput;
      else
        prd.throughput *= 1.f / p;
    }
  }

  // recursion did not terminate - cancel it
  return radiance;
}

// Remove NaN values
RT_FUNCTION float3 de_nan(const float3& c) {
  float3 temp = c;
  if (!(temp.x == temp.x)) temp.x = 0.f;
  if (!(temp.y == temp.y)) temp.y = 0.f;
  if (!(temp.z == temp.z)) temp.z = 0.f;

  return temp;
}

RT_FUNCTION uchar4 make_Color(float4 col) {
  float3 temp = sqrt(make_float3(col.x, col.y, col.z) / (frame + 1));

  int r = int(255.99 * Clamp(temp.x, 0.f, 1.f));  // R
  int g = int(255.99 * Clamp(temp.y, 0.f, 1.f));  // G
  int b = int(255.99 * Clamp(temp.z, 0.f, 1.f));  // B
  int a = int(255.99 * Clamp(1.f, 0.f, 1.f));     // A

  return make_uchar4(r, g, b, a);
}

RT_PROGRAM void renderPixel() {
  // get RNG seed
  uint seed = tea<16>(launchDim.x * pixelID.y + pixelID.x, frame);

  // initialize acc buffer if needed
  uint2 index = make_uint2(pixelID.x, launchDim.y - pixelID.y - 1);
  if (frame == 0) acc_buffer[index] = make_float4(0.f);

  // Subpixel jitter: send the ray through a different position inside the
  // pixel each time, to provide antialiasing.
  float u = float(pixelID.x + rnd(seed)) / float(launchDim.x);
  float v = float(pixelID.y + rnd(seed)) / float(launchDim.y);

  // trace ray
  Ray ray = Camera::generateRay(u, v, seed);

  // accumulate color
  float3 col = de_nan(color(ray, seed));
  acc_buffer[index] += make_float4(col.x, col.y, col.z, 1.f);
  display_buffer[index] = make_Color(acc_buffer[index]);
}
