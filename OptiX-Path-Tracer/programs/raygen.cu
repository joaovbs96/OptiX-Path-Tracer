#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

// optix code:
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "prd.h"
#include "sampling.h"

/*! the 'builtin' launch index we need to render a frame */
rtDeclareVariable(uint2, pixelID,   rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim,   );

/*! the ray related state */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData, prd, rtPayload, );

/*! the 2D, float3-type color frame buffer we'll write into */
rtBuffer<float3, 2> fb;

rtDeclareVariable(int, numSamples, , );
rtDeclareVariable(int, run, , );

rtDeclareVariable(rtObject, world, , );

rtDeclareVariable(int, light, , );

rtDeclareVariable(float3, camera_lower_left_corner, , );
rtDeclareVariable(float3, camera_horizontal, , );
rtDeclareVariable(float3, camera_vertical, , );
rtDeclareVariable(float3, camera_origin, , );
rtDeclareVariable(float3, camera_u, , );
rtDeclareVariable(float3, camera_v, , );
rtDeclareVariable(float, camera_lens_radius, , );
rtDeclareVariable(float, time0, , );
rtDeclareVariable(float, time1, , );

struct Camera {
  static __device__ optix::Ray generateRay(float s, float t, DRand48 &rnd) {
    const vec3f rd = camera_lens_radius * random_in_unit_disk(rnd);
    const vec3f lens_offset = camera_u * rd.x + camera_v * rd.y;
    const vec3f origin = camera_origin + lens_offset;
    const vec3f direction
      = camera_lower_left_corner
      + s * camera_horizontal
      + t * camera_vertical
      - origin;
    return optix::make_Ray(/* origin   : */ origin.as_float3(),
                           /* direction: */ direction.as_float3(),
                           /* ray type : */ 0,
                           /* tmin     : */ 1e-6f,
                           /* tmax     : */ RT_DEFAULT_MAX);
  }
};

inline __device__ vec3f missColor(const optix::Ray &ray) {
  if(light){
    const vec3f unit_direction = normalize(ray.direction);
    const float t = 0.5f*(unit_direction.y + 1.0f);
    const vec3f c = (1.0f - t) * vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
    return c;
  }
  else
    return vec3f(0.f);
}

inline __device__ vec3f color(optix::Ray &ray, DRand48 &rnd) {
  PerRayData prd;
  prd.in.randState = &rnd;
  prd.in.time = time0 + rnd() * (time1 - time0);

  vec3f attenuation = 1.f;
  
  /* iterative version of recursion, up to depth 50 */
  for (int depth = 0; depth < 50; depth++) {
    rtTrace(world, ray, prd);
    if (prd.out.scatterEvent == rayDidntHitAnything){
      // ray got 'lost' to the environment - 'light' it with miss shader
      return attenuation * missColor(ray);
    }

    else if (prd.out.scatterEvent == rayGotCancelled)
      return attenuation * prd.out.emitted;

    else { // ray is still alive, and got properly bounced
      attenuation = prd.out.emitted + attenuation * prd.out.attenuation;
      ray = optix::make_Ray(/* origin   : */ prd.out.scattered_origin.as_float3(),
                            /* direction: */ prd.out.scattered_direction.as_float3(),
                            /* ray type : */ 0,
                            /* tmin     : */ 1e-3f,
                            /* tmax     : */ RT_DEFAULT_MAX);
    }
  }
  // recursion did not terminate - cancel it
  return vec3f(0.f);
}

/*! the actual ray generation program - note this has no formal
  function parameters, but gets its paramters throught the 'pixelID'
  and 'pixelBuffer' variables/buffers declared above */
RT_PROGRAM void renderPixel() {
  int pixel_index = pixelID.y * launchDim.x + pixelID.x;
  vec3f col(0.f, 0.f, 0.f);
  DRand48 rnd;
  rnd.init(pixel_index + run * numSamples);

  for (int s = 0; s < numSamples; s++) {
    float u = float(pixelID.x + rnd()) / float(launchDim.x);
    float v = float(pixelID.y + rnd()) / float(launchDim.y);
    optix::Ray ray = Camera::generateRay(u, v, rnd);
    col += color(ray, rnd);
  }

  // the buffer keeps its previous state unless it's initialized again
  fb[pixelID] += col.as_float3();
}

