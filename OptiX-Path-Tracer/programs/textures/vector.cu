#include "hip/hip_runtime.h"
#include "texture.cuh"

rtDeclareVariable(int, size, , );
rtBuffer<rtCallableProgramId<float3(float, float, float3)> > texture_vector;

RT_CALLABLE_PROGRAM float3 sample_texture(float u, float v, float3 p,
                                          int index) {
  if (index >= size)
    return make_float3(0.f);
  else
    return texture_vector[index](u, v, p);
}